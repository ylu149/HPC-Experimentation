/****************************************************************************


   nvcc sor_cuda.cu -o sor_cuda 
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>

#ifdef __APPLE__
/* Shim for Mac OS X (use at your own risk ;-) */
# include "apple_pthread_barrier.h"
#endif /* __APPLE__ */

#define CPNS 2.0    /* Cycles per nanosecond -- Adjust to your computer,
                       for example a 3.2 GhZ GPU, this would be 3.2 */
#define A   0   /* coefficient of x^2 */
#define B   0  /* coefficient of x */
#define C   2048 /* constant term */

#define NUM_TESTS 1
#define bl_size 16
#define ITERS 2000

/* A, B, and C needs to be a multiple of your BLOCK_SIZE,
   total array size will be ( + Ax^2 + Bx + C) */
#define MINVAL   0.0
#define MAXVAL  10.0
#define TOL 0.00001

typedef double data_t;

typedef struct {
  long int rowlen;
  data_t *data;
} arr_rec, *arr_ptr;

/* Prototypes */
arr_ptr new_array(long int row_len);
int set_arr_rowlen(arr_ptr v, long int index);
long int get_arr_rowlen(arr_ptr v);
int init_array(arr_ptr v, long int row_len);
int init_array_rand(arr_ptr v, long int row_len);
void print_array(arr_ptr v);
data_t *get_array_start(arr_ptr v);
__global__ void gpu_SOR(data_t *data , data_t *interations, long int rowlen);

void SOR(arr_ptr v, int *iterations);

/* -=-=-=-=- Time measurement by clock_gettime() -=-=-=-=- */
/*
  As described in the clock_gettime manpage (type "man clock_gettime" at the
  shell prompt), a "timespec" is a structure that looks like this:
 
        struct timespec {
          time_t   tv_sec;   // seconds
          long     tv_nsec;  // and nanoseconds
        };
 */

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}
/*
     This method does not require adjusting a #define constant

  How to use this method:

      struct timespec time_start, time_stop;
      clock_gettime(CLOCK_REALTIME, &time_start);
      // DO SOMETHING THAT TAKES TIME
      clock_gettime(CLOCK_REALTIME, &time_stop);
      measurement = interval(time_start, time_stop);

 */


/* -=-=-=-=- End of time measurement declarations =-=-=-=- */

/*****************************************************************************/
int main(int argc, char *argv[])
{
  struct timespec time_start, time_stop;
  double time_stamp[NUM_TESTS];
  int convergence[NUM_TESTS];
  int *iterations;
  long int x, n;
  long int alloc_size;

  x = NUM_TESTS-1;
  alloc_size =  A*x*x + B*x + C;

  printf("SOR serial code \n");
  /* declare and initialize the array */
  arr_ptr v0 = new_array(alloc_size);
  
  /* Allocate space for return value */
  iterations = (int *) malloc(sizeof(int));
  init_array_rand(v0, alloc_size);
  set_arr_rowlen(v0, alloc_size);
  
  /* Prep cuda code*/
  data_t *d_arrptr, *d_iterations;
  if(hipMalloc(&d_arrptr, sizeof(data_t) * alloc_size*alloc_size) != hipSuccess){
      std::cout<<("Cannot allocate device memory.");
      return 0;
  } 
  if(hipMalloc(&d_iterations, sizeof(data_t) * alloc_size*alloc_size) != hipSuccess){
      std::cout<<("Cannot allocate device memory.");
      free(d_arrptr);
      return 0;
  }
  if(hipMemcpy(d_arrptr, v0->data, sizeof(data_t) * alloc_size*alloc_size, hipMemcpyHostToDevice) != hipSuccess){
      std::cout<<("Cannot copy from host to device.");
      free(d_arrptr);
      free(d_iterations);
      return 0;
  } 
  
  /* initialize cta and grid dimensions*/
  int threads = bl_size;
  int blocks = 1;
  
  //setup kernal launch parameters
  dim3 THREADS(threads,threads);
  dim3 BLOCKS(blocks, blocks);
  
  //std::cout<<"original v0\n";
  //print_array(v0);
  //std::cout<<std::endl;
  //gpu_SOR<<<BLOCKS, THREADS>>>(d_arrptr , d_iterations, alloc_size);
  
   /*if(cudaMemcpy(v0->data, d_arrptr, sizeof(data_t) * alloc_size*alloc_size, cudaMemcpyDeviceToHost) != cudaSuccess){
      std::cout<<("Cannot copy from host to device.");
      free(d_arrptr);
      free(d_iterations);
      return 0;
  }*/
  //std::cout<<"after gpu_SOR\n";
  //print_array(v0);
  
  for (x=0; x<NUM_TESTS && (n = A*x*x + B*x + C, n<=alloc_size); x++) {
    //printf("  iter %ld rowlen = %ld\n", x, n);
    init_array_rand(v0, n);
    set_arr_rowlen(v0, n);
    clock_gettime(CLOCK_REALTIME, &time_start);
    gpu_SOR<<<BLOCKS, THREADS>>>(d_arrptr, d_iterations, alloc_size);
    clock_gettime(CLOCK_REALTIME, &time_stop);
    if(hipMemcpy(v0->data, d_arrptr, sizeof(data_t) * alloc_size*alloc_size, hipMemcpyDeviceToHost) != hipSuccess){
      std::cout<<("Cannot copy from host to device.");
      free(d_arrptr);
      free(d_iterations);
      return 0;
      }
    time_stamp[x] = interval(time_start, time_stop);
    convergence[x] = *iterations;
  }

  printf("\n");
  printf("#Elements, SOR time\n");
  {
    int i;
    for (i = 0; i < NUM_TESTS; i++) {
      printf("%4d", A*i*i + B*i + C);
      printf(", %10.4g",time_stamp[i]);
      //printf(", %4d", convergence[i]);
      printf("\n");
    }
  }

} /* end main */

/*********************************/

/* Create 2D array of specified length per dimension */
arr_ptr new_array(long int row_len)
{
  /* Allocate and declare header structure */
  arr_ptr result = (arr_ptr) malloc(sizeof(arr_rec));
  if (!result) {
    return NULL;  /* Couldn't allocate storage */
  }
  result->rowlen = row_len;

  /* Allocate and declare array */
  if (row_len > 0) {
    data_t *data = (data_t *) calloc(row_len*row_len, sizeof(data_t));
    if (!data) {
      free((void *) result);
      printf("\n COULDN'T ALLOCATE STORAGE \n");
      return NULL;  /* Couldn't allocate storage */
    }
    result->data = data;
  }
  else result->data = NULL;

  return result;
}

/* Set row length of array */
int set_arr_rowlen(arr_ptr v, long int row_len)
{
  v->rowlen = row_len;
  return 1;
}

/* Return row length of array */
long int get_arr_rowlen(arr_ptr v)
{
  return v->rowlen;
}

/* initialize 2D array with incrementing values (0.0, 1.0, 2.0, 3.0, ...) */
int init_array(arr_ptr v, long int row_len)
{
  long int i;

  if (row_len > 0) {
    v->rowlen = row_len;
    for (i = 0; i < row_len*row_len; i++) {
      v->data[i] = (data_t)(i);
    }
    return 1;
  }
  else return 0;
}

/* initialize array with random data */
int init_array_rand(arr_ptr v, long int row_len)
{
  long int i;
  double fRand(double fMin, double fMax);

  /* Since we're comparing different algorithms (e.g. blocked, threaded
     with stripes, red/black, ...), it is more useful to have the same
     randomness for any given array size */
  srandom(row_len);
  if (row_len > 0) {
    v->rowlen = row_len;
    for (i = 0; i < row_len*row_len; i++) {
      v->data[i] = (data_t)(fRand((double)(MINVAL),(double)(MAXVAL)));
    }
    return 1;
  }
  else return 0;
}

/* print all elements of an array */
void print_array(arr_ptr v)
{
  long int i, j, row_len;

  row_len = v->rowlen;
  printf("row length = %ld\n", row_len);
  for (i = 0; i < row_len; i++) {
    for (j = 0; j < row_len; j++) {
      printf("%.4f ", (data_t)(v->data[i*row_len+j]));
    }
    printf("\n");
  }
}

data_t *get_array_start(arr_ptr v)
{
  return v->data;
}

double fRand(double fMin, double fMax)
{
  double f = (double)random() / RAND_MAX;
  return fMin + f * (fMax - fMin);
}

/************************************/

/* SOR */
void SOR(arr_ptr v, int *iterations)
{
  long int i, j;
  long int rowlen = get_arr_rowlen(v);
  data_t *data = get_array_start(v);
  double diff = 1.0e10;   /* start w/ something big */
  int iters = 0, temp = 0;

  while ((diff/(double)(rowlen*rowlen)) > (double)TOL) {
    iters++;
    diff = 0;
    for (i = 1; i < rowlen-1; i++) {
      for (j = 1; j < rowlen-1; j++) {
        temp = data[i*rowlen+j];
        data[i*rowlen+j] =         0.2 * (  data[(i)*rowlen+j] +
                                          data[(i)*rowlen+(j-1)] +
                                          data[(i-1)*rowlen+j] +
                                          data[i*rowlen+(j+1)]   +
                                          data[(i+1)*rowlen+j]);
        diff += abs(data[i*rowlen+j] - temp);
      }
    }
    if (abs(data[(rowlen-2)*(rowlen-2)]) > 10.0*(MAXVAL - MINVAL)) {
      printf("SOR: SUSPECT DIVERGENCE iter = %d\n", iters);
      break;
    }
  }
  *iterations = iters;
  //printf("    SOR() done after %d iters\n", iters);
}

__global__ void gpu_SOR(data_t *data , data_t *iterations, long int rowlen)
{
  long int i = blockIdx.y * blockDim.y + threadIdx.y; //row
  long  int j = blockIdx.x * blockDim.x + threadIdx.x; //col
  long int x, y, iters;
  int a, b;
  data_t arr[bl_size*bl_size];

  //boundary check
  if(i < rowlen && j < rowlen && i>= 0 && j>=0){
     for (iters = 0; iters < ITERS; iters++){ 
        for(x = i*bl_size; x < i*bl_size+bl_size; x++){
            for(y = j*bl_size;  y < j*bl_size+bl_size; y++){
               a = x - (i*bl_size);
               b = y - (j*bl_size); 
               if(x > 0 && y > 0 && x < bl_size-1 && y < bl_size-1){ 
                     arr[a*bl_size+b] =         0.2 * (  data[(x)*rowlen+y] +
                                          data[(x)*rowlen+(y-1)] +
                                          data[(x-1)*rowlen+y] +
                                          data[x*rowlen+(j+y)]   +
                                          data[(x+1)*rowlen+y]);
               }
               else{
                  arr[a*bl_size+b] = data[(x)*rowlen+y];  
               }
            }
        }
    
        __syncthreads();
        for(x = i*bl_size; x < i*bl_size+bl_size; x++){
            for(y = j*bl_size;  y < j*bl_size+bl_size; y++){
               a = x - (i*bl_size);
               b = y - (j*bl_size);
               data[x*rowlen+y] = arr[a*bl_size+b];
            }
        }
        __syncthreads();
     }
  }
}
