/*****************************************************************************/
// nvcc hw8.cu -o hw8

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <iostream>

/* We want to test a wide range of work sizes. We will generate these
   using the quadratic formula:  A x^2 + B x + C                     */
#define A   0  /* coefficient of x^2 */
#define B   0  /* coefficient of x */
#define C   2048  /* constant term */

#define NUM_TESTS 1   /* Number of different sizes to test */

#define OPTIONS 2
#define IDENT 0

typedef float data_t;

/* Create abstract data type for matrix */
typedef struct {
  long int len;
  data_t *data;
} matrix_rec, *matrix_ptr;

/* Prototypes */
int clock_gettime(clockid_t clk_id, struct timespec *tp);
matrix_ptr new_matrix(long int row_len);
int set_matrix_row_length(matrix_ptr m, long int row_len);
long int get_matrix_row_length(matrix_ptr m);
int init_matrix(matrix_ptr m, long int row_len);
int zero_matrix(matrix_ptr m, long int row_len);
void mmm_ijk(matrix_ptr a, matrix_ptr b, matrix_ptr c);
bool MMM_check(matrix_ptr expected, matrix_ptr actual);
__global__ void gpu_MMM(data_t *da0, data_t *db0, data_t *dc0, long int rowlen);
void printMat(matrix_ptr mat, long int rowlen);

/* -=-=-=-=- Time measurement by clock_gettime() -=-=-=-=- */
/*
  As described in the clock_gettime manpage (type "man clock_gettime" at the
  shell prompt), a "timespec" is a structure that looks like this:
 
        struct timespec {
          time_t   tv_sec;   // seconds
          long     tv_nsec;  // and nanoseconds
        };
 */

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}
/*
     This method does not require adjusting a #define constant

  How to use this method:

      struct timespec time_start, time_stop;
      clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
      // DO SOMETHING THAT TAKES TIME
      clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
      measurement = interval(time_start, time_stop);

 */


/* -=-=-=-=- End of time measurement declarations =-=-=-=- */

/* This routine "wastes" a little time to make sure the machine gets
   out of power-saving mode (800 MHz) and switches to normal speed. */
double wakeup_delay()
{
  srand(time(0));
  double meas = 0; int i, j;
  struct timespec time_start, time_stop;
  double quasi_random = 0;
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
  j = 100;
  while (meas < 1.0) {
    for (i=1; i<j; i++) {
      /* This iterative calculation uses a chaotic map function, specifically
         the complex quadratic map (as in Julia and Mandelbrot sets), which is
         unpredictable enough to prevent compiler optimisation. */
      quasi_random = quasi_random*quasi_random - 1.923432;
    }
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
    meas = interval(time_start, time_stop);
    j *= 2; /* Twice as much delay next time, until we've taken 1 second */
  }
  return quasi_random;
}

/*****************************************************************************/
int main(int argc, char *argv[])
{
  srand(time(0));
  int OPTION;
  struct timespec time_start, time_stop, gpu_timestart, gpu_timestop;
  double time_stamp[OPTIONS][NUM_TESTS];
  double wakeup_answer;
  long int x, n, alloc_size;

  x = NUM_TESTS-1;
  alloc_size = A*x*x + B*x + C;

  printf("Dense MMM tests \n\n");

  wakeup_answer = wakeup_delay();

  printf("Doing MMM three different ways,\n");
  printf("for %d different matrix sizes from %d to %ld\n", NUM_TESTS, C, alloc_size);
  printf("This may take a while!\n\n");
  /* declare and initialize the matrix structure */
  matrix_ptr a0 = new_matrix(alloc_size);
  init_matrix(a0, alloc_size);
  matrix_ptr b0 = new_matrix(alloc_size);
  init_matrix(b0, alloc_size);
  matrix_ptr c0 = new_matrix(alloc_size);
  zero_matrix(c0, alloc_size);
  matrix_ptr test_gpu = new_matrix(alloc_size);
  zero_matrix(test_gpu, alloc_size);
 
  OPTION = 0;//serial
  for (x=0; x<NUM_TESTS && (n = A*x*x + B*x + C, n<=alloc_size); x++) {
    set_matrix_row_length(a0, n);
    set_matrix_row_length(b0, n);
    set_matrix_row_length(c0, n);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
    mmm_ijk(a0, b0, c0);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
    time_stamp[OPTION][x] = interval(time_start, time_stop);
  }
   
  //initialize GPU
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &gpu_timestart);
  data_t *da0, *db0, *dc0;
  long int size = alloc_size*alloc_size;
  if(hipMalloc(&da0, sizeof(data_t)*size) != hipSuccess);
  if(hipMalloc(&db0, sizeof(data_t)*size) != hipSuccess);
  if(hipMalloc(&dc0, sizeof(data_t)*size) != hipSuccess);
  if(hipMemcpy(da0, a0->data, sizeof(data_t)*size, hipMemcpyHostToDevice) != hipSuccess);
  if(hipMemcpy(db0, b0->data, sizeof(data_t)*size, hipMemcpyHostToDevice) != hipSuccess);
  if(hipMemcpy(dc0, c0->data, sizeof(data_t)*size, hipMemcpyHostToDevice) != hipSuccess);  
  dim3 BLOCK(16, 16, 1);
  dim3 GRID(size/16, size/16, 1);
  OPTION++;//GPU
  for (x=0; x<NUM_TESTS && (n = A*x*x + B*x + C, n<=alloc_size); x++) {
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_start);
    gpu_MMM<<<GRID, BLOCK>>>(da0, db0, dc0, alloc_size);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time_stop);
    time_stamp[OPTION][x] = interval(time_start, time_stop);
    //checking for errors by comparing serial vs GPU
    set_matrix_row_length(test_gpu, alloc_size);
    if(hipMemcpy(test_gpu->data, dc0, sizeof(data_t)*size, hipMemcpyDeviceToHost) != hipSuccess){std::cout<<"fail\n"; return 0;}
    printf("\nTrue/False: %d\n",MMM_check(c0, test_gpu));
    //printMat(c0, n);
    //printMat(test_gpu, n);
  }
  clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &gpu_timestop);
  
  //printf("Done collecting measurements.\n\n");
  printf("Total gpu overhead: %f\n\n", interval(gpu_timestart, gpu_timestop));
  printf("row_len, ijk_cpu, GPU_Serial\n");
  {
    int i, j;
    for (i = 0; i < NUM_TESTS; i++) {
      printf("%d, ", A*i*i + B*i + C);
      for (j = 0; j < OPTIONS; j++) {
        if (j != 0) {
          printf(", ");
        }
        printf("%f", time_stamp[j][i]);
      }
      printf("\n");
    }
  }
  printf("\n");

  printf("Wakeup delay computed: %g \n", wakeup_answer);
} /* end main */

/**********************************************/

/* Create matrix of specified length */
matrix_ptr new_matrix(long int row_len)
{
  //long int i;
  long int alloc;

  /* Allocate and declare header structure */
  matrix_ptr result = (matrix_ptr) malloc(sizeof(matrix_rec));
  if (!result) return NULL;  /* Couldn't allocate storage */
  result->len = row_len;

  /* Allocate and declare array */
  if (row_len > 0) {
    alloc = row_len * row_len;
    data_t *data = (data_t *) calloc(alloc, sizeof(data_t));
    if (!data) {
	  free((void *) result);
	  printf("\n COULDN'T ALLOCATE %ld BYTES STORAGE \n",
                                                       alloc * sizeof(data_t));
	  return NULL;  /* Couldn't allocate storage */
	}
	result->data = data;
  } else {
    result->data = NULL;
  }

  return result;
}

/* Set length of matrix */
int set_matrix_row_length(matrix_ptr m, long int row_len)
{
  m->len = row_len;
  return 1;
}

/* Return length of matrix */
long int get_matrix_row_length(matrix_ptr m)
{
  return m->len;
}

/* initialize matrix */
int init_matrix(matrix_ptr m, long int row_len)
{
  long int i;

  if (row_len > 0) {
    m->len = row_len;
    for (i = 0; i < row_len*row_len; i++) {
      m->data[i] = (data_t)(rand() % 100 + 0);
    }
    return 1;
  }
  else return 0;
}

/* initialize matrix */
int zero_matrix(matrix_ptr m, long int row_len)
{
  long int i;

  if (row_len > 0) {
    m->len = row_len;
    for (i = 0; i < row_len*row_len; i++) {
      m->data[i] = (data_t)(IDENT);
    }
    return 1;
  }
  else return 0;
}

data_t *get_matrix_start(matrix_ptr m)
{
  return m->data;
}

/*************************************************/

/* mmm */
void mmm_ijk(matrix_ptr a, matrix_ptr b, matrix_ptr c)
{
  long int i, j, k;
  long int length = get_matrix_row_length(a);
  data_t *a0 = get_matrix_start(a);
  data_t *b0 = get_matrix_start(b);
  data_t *c0 = get_matrix_start(c);
  data_t sum;

  for (i = 0; i < length; i++) {
    for (j = 0; j < length; j++) {
      sum = IDENT;
      for (k = 0; k < length; k++) {
        sum += a0[i*length+k] * b0[k*length+j];
      }
      c0[i*length+j] += sum;
    }
  }
}

void printMat(matrix_ptr mat, long int rowlen)
{
   data_t *arr = get_matrix_start(mat);
   long int i, j;
    printf("\n");
    for (i = 0; i < rowlen; i++){
        for(j = 0; j < rowlen; j++){
            printf("%0.2f ", arr[i*rowlen+j]);
        }
        printf("\n");
    }
}
bool MMM_check(matrix_ptr expected, matrix_ptr actual)
{
    data_t *serial = get_matrix_start(expected);
    data_t *gpu = get_matrix_start(actual);
    long int length = get_matrix_row_length(expected);
    long int i, j;
    long int imax = 0, jmax = 0;
    long int max_delta=0;
    for (i = 0; i < length; i++){
        for(j = 0; j < length; j++){
            if ((long int)serial[i*length+j] != (long int)gpu[i*length+j]){
                //printf("\nserial: %f GPU: %f\nRow: %ld Col: %ld\n", serial[i*length+j], gpu[i*length+j],i,j);
                //return false;
               if(max_delta < abs(serial[i*length+j]-gpu[i*length+j])){ 
                   max_delta = abs(serial[i*length+j]-gpu[i*length+j]);
                   imax = i;
                   jmax = j;
               }
               //return false;
            }
        }
    }
    printf("Biggest difference:\nserial: %f, gpu: %f\n\n", serial[imax*length+jmax], gpu[imax*length+jmax]);
    return true;
}

__global__ void gpu_MMM(data_t *da0, data_t *db0, data_t *dc0, long int rowlen)
{
    long int n = rowlen*rowlen;
    int nElem = n/16;
    int b_incrament = n*16;
    int aStart = (16*blockIdx.y+threadIdx.y)*n+threadIdx.x;
    int bStart = 16*blockIdx.x+threadIdx.y*n+threadIdx.x;
    __shared__ data_t a[16][16], b[16][17];
    data_t sum = 0;
    long int i;

    for (i = 0; i < nElem; i++){
        a[threadIdx.y][threadIdx.x] = da0[aStart];
        b[threadIdx.x][threadIdx.y] = db0[bStart];
        __syncthreads();
        //for (k = 0; k < 16; k+=4)
        {
            sum += a[threadIdx.y][0]*b[threadIdx.x][0];
            sum += a[threadIdx.y][1]*b[threadIdx.x][1];
            sum += a[threadIdx.y][2]*b[threadIdx.x][2];
            sum += a[threadIdx.y][3]*b[threadIdx.x][3];
            sum += a[threadIdx.y][4]*b[threadIdx.x][4];
            sum += a[threadIdx.y][5]*b[threadIdx.x][5];
            sum += a[threadIdx.y][6]*b[threadIdx.x][6];
            sum += a[threadIdx.y][7]*b[threadIdx.x][7];
            sum += a[threadIdx.y][8]*b[threadIdx.x][8];
            sum += a[threadIdx.y][9]*b[threadIdx.x][9];
            sum += a[threadIdx.y][10]*b[threadIdx.x][10];
            sum += a[threadIdx.y][11]*b[threadIdx.x][11];
            sum += a[threadIdx.y][12]*b[threadIdx.x][12];
            sum += a[threadIdx.y][13]*b[threadIdx.x][13];
            sum += a[threadIdx.y][14]*b[threadIdx.x][14];
            sum += a[threadIdx.y][15]*b[threadIdx.x][15];
 
        }
        //__syncthreads();
        aStart += 16;
        bStart += b_incrament;
    }
    dc0[(16*blockIdx.y+threadIdx.y)*n + 16*blockIdx.x + threadIdx.x] = sum;
}
